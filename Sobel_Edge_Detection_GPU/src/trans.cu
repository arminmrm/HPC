
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>
#include <math.h>
#define TIMER_CREATE(t)               \
  hipEvent_t t##_start, t##_end;     \
  hipEventCreate(&t##_start);        \
  hipEventCreate(&t##_end);               
 
 
#define TIMER_START(t)                \
  hipEventRecord(t##_start);         \
  hipEventSynchronize(t##_start);    \
 
 
#define TIMER_END(t)                             \
  hipEventRecord(t##_end);                      \
  hipEventSynchronize(t##_end);                 \
  hipEventElapsedTime(&t, t##_start, t##_end);  \
  hipEventDestroy(t##_start);                   \
  hipEventDestroy(t##_end);     
  
#define TILE_SIZE 16
#define CUDA_TIMING

unsigned char *input_gpu;
unsigned char *output_gpu;

/*******************************************************/
/*                 Cuda Error Function                 */
/*******************************************************/
inline hipError_t checkCuda(hipError_t result) {
    #if defined(DEBUG) || defined(_DEBUG)
    if (result != cudaSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", cudaGetErrorString(result));
        exit(-1);
    }
    #endif
    return result;
}
                
// GPU kernel and functions
__global__ void kernel(unsigned char *input, 
                       unsigned char *output,
                       unsigned int height,
                       unsigned int width){
        
    int x = blockIdx.x*TILE_SIZE+threadIdx.x;
    int y = blockIdx.y*TILE_SIZE+threadIdx.y;
    
    int i,j; 
    int sumx,sumy;
    unsigned long int sum;
    int hx[9] = {1,0,-1,2,0,-2,1,0,-1};
    int hy[9] = {1,2,1,0,0,0,-1,-2,-1};
    
    if (x < width && y < height ){ 
        sumx=0;
        sumy=0;
        for(j=x-1;j<x+2;j++)
            for(i=y-1;i<y+2;i++)
                if(j>=0 && j<width && i>=0 && i<height)
                {
                    sumx += input[i*width+j]*hx[(y-i+1)*3+x-j+1];
                    sumy += input[i*width+j]*hy[(y-i+1)*3+x-j+1];
                }
    }
    
    sum = sumx*sumx+sumy*sumy;

  
    if(sum>100)
        output[y*width+x] = 255;
    else
        output[y*width+x] = 0;    
}

void transpose_img(unsigned char *in_mat, 
                   unsigned char *out_mat, 
                   unsigned int height, 
                   unsigned int width){
                         
    int gridXSize = 1 + (( width - 1) / TILE_SIZE);
    int gridYSize = 1 + ((height - 1) / TILE_SIZE);
    
    int XSize = gridXSize*TILE_SIZE;
    int YSize = gridYSize*TILE_SIZE;
    
    // Both are the same size (CPU/GPU).
    int size = XSize*YSize;
    
    // Allocate arrays in GPU memory
    checkCuda(hipMalloc((void**)&input_gpu    , size*sizeof(unsigned char)));
    checkCuda(hipMalloc((void**)&output_gpu  , size*sizeof(unsigned char)));
    
    checkCuda(hipMemset(output_gpu , 0 , size*sizeof(unsigned char)));
                
    // Copy data to GPU
    checkCuda(hipMemcpy(input_gpu, 
                        in_mat, 
                        height*width*sizeof(char), 
                        hipMemcpyHostToDevice));

    checkCuda(hipDeviceSynchronize());
    
    // Execute algorithm
    dim3 dimGrid(gridXSize, gridYSize);
    dim3 dimBlock(TILE_SIZE, TILE_SIZE);

    #if defined(CUDA_TIMING)
        float Ktime;
        TIMER_CREATE(Ktime);
        TIMER_START(Ktime);
    #endif
    
    // Kernel Call
    kernel<<<dimGrid, dimBlock>>>(input_gpu, output_gpu, height, width);
    
    checkCuda(hipDeviceSynchronize());
    
    #if defined(CUDA_TIMING)
        TIMER_END(Ktime);
        printf("Kernel Execution Time: %f ms\n", Ktime);
    #endif
        
    // Retrieve results from the GPU
    checkCuda(hipMemcpy(out_mat, 
                        output_gpu, 
                        height*width*sizeof(unsigned char), 
                        hipMemcpyDeviceToHost));
                        
    // Free resources and end the program
    checkCuda(hipFree(output_gpu));
    checkCuda(hipFree(input_gpu));

}
